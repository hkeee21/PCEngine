#include "hip/hip_runtime.h"
#include "spconv.h"

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <torch/extension.h>

using namespace std;

#define BLOCK_SIZE 32


extern "C"

#define checkCudaError( a ) do { \
    if (hipSuccess != (a)) { \
    fprintf(stderr, "Cuda runTime error in line %d of file %s \
    : %s \n", __LINE__, __FILE__, hipGetErrorString(hipGetLastError()) ); \
    exit(EXIT_FAILURE); \
    } \
} while(0)


__global__ void center_map(const int nnz, int *map)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // a thread for a coord
    while(id < nnz)
    {
        map[id] = id;

        id += blockDim.x * gridDim.x;
    }

}


__global__ void gemm(const int nnz, const int kernel_nnz, const int c_in, const int c_out,
                const float *__restrict__ in_f, const float *__restrict__ kv, float *out_f,
                const long *nnz_idx, const int *map) {

  // Block index
  const int bx = blockIdx.x;
  const int by = blockIdx.y;

  // Thread index
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;

  // Coordinate. x is for rows, y is for columns.
  const int x = BLOCK_SIZE * bx + tx;
  const int y = BLOCK_SIZE * by + ty;
  // const int y = BLOCK_SIZE * bx + tx;
  // const int x = BLOCK_SIZE * by + ty;

  // The thread deals with the x-th channel of the y-th output
  const int out_row = y < kernel_nnz ? nnz_idx[y] % nnz : -1;
  const int in_row = y < kernel_nnz ? map[out_row] : -1;

  if(in_row > -1 && out_row > -1){
  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  float Csub = 0;
  
  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int s = 0; s < c_in; s += BLOCK_SIZE) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[ty][tx] = ((s + tx) < c_in && in_row < nnz) ? in_f[c_in * in_row + s + tx] : 0;
    Bs[ty][tx] = ((s + ty) < c_in && x < c_out) ? kv[c_out * (s + ty) + x] : 0;

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll
    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[ty][k] * Bs[k][tx];
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  if (out_row < nnz && x < c_out)
    atomicAdd(&out_f[c_out * out_row + x], Csub);
  // C[wB * out_row + x] += Csub;
  }
}


void ConvolutionForward(const at::Tensor in_coords, const at::Tensor in_feats, 
                        const at::Tensor kernel, const int k_size, 
                        const at::Tensor in_map, at::Tensor out_feats,
                        const at::Tensor kernel_nnz, const at::Tensor whole_idx
                        ){
    
    // printf("[SubmanifoldSparseConv] - Starts.\n");

    int nnz = in_coords.size(0);
    int in_channel = in_feats.size(1);
    if (in_feats.size(1) != kernel.size(1)) {
        throw std::invalid_argument("Input feature size and kernel size mismatch");
    }
    int out_channel = kernel.size(2);
    int k_vol = k_size * k_size * k_size;

    float *in_feats_ptr = in_feats.data_ptr<float>();
    float *weight_ptr = kernel.data_ptr<float>();
    float *out_feats_ptr = out_feats.data_ptr<float>();
    int *in_coords_ptr = in_coords.data_ptr<int>();
    int *in_map_ptr = in_map.data_ptr<int>();
    long *whole_idx_ptr = whole_idx.data_ptr<long>();


    // loop over all kernel offsets
    int cur_idx = 0;

    // Suppose an odd kernel size
    for (int i = 0; i < k_vol; i++){

        // calculate the kernel offset
        /*int k_offset_x = i / (k_size * k_size) - (k_size - 1) / 2;
        int k_offset_y = (i / k_size) % k_size - (k_size - 1) / 2;
        int k_offset_z = i % k_size - (k_size - 1) / 2;

        // search the nnz involved and record the mapping
        // kernel offset (0, 0, 0) need no mapping calculation

        if (k_offset_x == 0 && k_offset_y == 0 && k_offset_z == 0){
            center_map<<<dim3(blocknum, 1, 1), dim3(BLOCK_SIZE, 1, 1)>>>(nnz, in_map_ptr);
        }
        else{
            search<<<dim3(blocknum, 1, 1), dim3(BLOCK_SIZE, 1, 1)>>>(
                nnz, 
                in_coords_ptr, 
                k_size, 
                k_offset_x, k_offset_y, k_offset_z, 
                in_map_ptr);
            
            queryHash<<<dim3(blocknum, 1, 1), dim3(BLOCK_SIZE, 1, 1)>>>(
                nnz, 
                table_size, 
                in_coords_ptr,
                k_size, 
                k_offset_x, 
                k_offset_y, 
                k_offset_z, 
                value_ptr, 
                index_ptr, 
                in_map_ptr
                );
        }
        
        at::Tensor kernel_map;
        kernel_map = torch::from_blob(&in_map_ptr[i * nnz], {nnz}, at::device(in_map.device()).dtype(at::ScalarType::Int));
        at::Tensor nnz_idx = torch::nonzero(kernel_map + torch::ones_like(kernel_map));  // torch::nonzero returns long tensor
        int kernel_nnz = nnz_idx.size(0);*/

        int cur_nnz = kernel_nnz[i].item<int>();

        if (cur_nnz == 0){continue;}

        // size_t const gridnum_x = (out_channel + BLOCK_SIZE - 1) / BLOCK_SIZE;
        // size_t const gridnum_y = (kernel_nnz + BLOCK_SIZE - 1) / BLOCK_SIZE;
        size_t const gridnum_x = (out_channel + BLOCK_SIZE - 1) / BLOCK_SIZE;
        size_t const gridnum_y = (cur_nnz + BLOCK_SIZE - 1) / BLOCK_SIZE;

        // GEMM
        gemm<<<dim3(gridnum_x, gridnum_y, 1), dim3(BLOCK_SIZE, BLOCK_SIZE, 1)>>>(
                nnz, 
                cur_nnz, 
                in_channel, out_channel,
                in_feats_ptr,
                &weight_ptr[i * in_channel * out_channel],
                out_feats_ptr,
                &whole_idx_ptr[cur_idx], 
                &in_map_ptr[i * nnz]);
        
        cur_idx += cur_nnz;
    
    }

    // printf("[SubmanifoldSparseConv] - Ends.\n");

    hipError_t err = hipGetLastError();
    if (err != hipSuccess){
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

}
