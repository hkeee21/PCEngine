#include "spconv.cuh"
#include "spconv.h"

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "cublas_utils.h"

#define DIV_UP(x, y) (x + y - 1) / y

extern "C"

    void
    ConvolutionForward(at::Tensor in_feats, at::Tensor kernel,
                       const int ksize_code, const int qsum_nnz,
                       at::Tensor out_feats, const at::Tensor kernel_kpos,
                       const at::Tensor kernel_qkpos, const at::Tensor in_map,
                       const at::Tensor out_map, const at::Tensor in_csr,
                       const at::Tensor out_csr, at::Tensor buffer,
                       const bool separate_mid, const bool TensorCore16Fast) {

  // printf("[SubmanifoldSparseConv] - Starts.\n");

  int in_nnz = in_feats.size(0);
  int out_nnz = out_feats.size(0);
  int in_channel = in_feats.size(1);
  int in_channel_mod = in_channel;
  if (in_feats.size(1) != kernel.size(1)) {
    throw std::invalid_argument("Input feature size and kernel size mismatch");
  }
  int out_channel = kernel.size(2);
  int k_vol = kernel.size(0);
  int one_k_offset = in_channel * out_channel;

  bool data_type_half = in_feats.scalar_type() == at::ScalarType::Half;

  int *in_map_ptr = in_map.data_ptr<int>();
  int *out_map_ptr = out_map.data_ptr<int>();
  int *in_csr_ptr = in_csr.data_ptr<int>();
  int *out_csr_ptr = out_csr.data_ptr<int>();
  int *kpos_ptr = kernel_kpos.data_ptr<int>();
  int *qkpos_ptr = kernel_qkpos.data_ptr<int>();

  // int sum_nnz = in_buffer.size(0);
  // printf("sum nnz: %d", sum_nnz);

  int ksx = ksize_code / 311;
  int ksy = (ksize_code - ksx * 311) / 17;
  int ksz = ksize_code - ksx * 311 - ksy * 17;
  int mid_weight_id =
      (ksx - 1) / 2 * ksy * ksz + (ksy - 1) / 2 * ksz + (ksz - 1) / 2;

  // cublas
  const float alpha = 1.0;
  const float alpha_copy = 1.0;
  const float beta = 0.0;
  at::Tensor alpha_half = torch::ones({1}, dtype(at::ScalarType::Half));
  at::Tensor alpha_copy_half = torch::ones({1}, dtype(at::ScalarType::Half));
  at::Tensor beta_half = torch::zeros({1}, dtype(at::ScalarType::Half));

  hipblasHandle_t cublasH = at::cuda::getCurrentCUDABlasHandle();

  CUBLAS_CHECK(hipblasSetMathMode(cublasH, HIPBLAS_TENSOR_OP_MATH));

  hipblasComputeType_t ComputeType;
  hipDataType DataType;
  if (data_type_half) {
    ComputeType = HIPBLAS_COMPUTE_16F;
    DataType = HIP_R_16F;
  } else {
    ComputeType = TensorCore16Fast ? HIPBLAS_COMPUTE_32F_FAST_16F
                                   : HIPBLAS_COMPUTE_32F_FAST_TF32;
    DataType = HIP_R_32F;
  }

  /********************************************************************/
  // default stream

  if (data_type_half) {
    if (in_channel % 4 == 0) {
      gather_all_input_major_csr_half_4<<<
          DIV_UP(in_nnz, 4), dim3(DIV_UP(in_channel, 4), 2, 4), 0, 0>>>(
          in_nnz, in_channel,
          reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()), qkpos_ptr,
          in_csr_ptr, in_map_ptr,
          reinterpret_cast<half *>(buffer.data_ptr<at::Half>()));
    } else {
      gather_all_input_major_csr_half_2<<<
          DIV_UP(in_nnz, 4), dim3(DIV_UP(in_channel, 2), 2, 4), 0, 0>>>(
          in_nnz, in_channel,
          reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()), qkpos_ptr,
          in_csr_ptr, in_map_ptr,
          reinterpret_cast<half *>(buffer.data_ptr<at::Half>()));
    }
  } else {
    if (in_channel % 4 == 0) {
      gather_all_input_major_csr_float_4<<<
          DIV_UP(in_nnz, 4), dim3(DIV_UP(in_channel, 4), 2, 4), 0, 0>>>(
          in_nnz, in_channel, in_feats.data_ptr<float>(), qkpos_ptr, in_csr_ptr,
          in_map_ptr, buffer.data_ptr<float>());
    } else {
      gather_all_input_major_csr_float_2<<<
          DIV_UP(in_nnz, 4), dim3(DIV_UP(in_channel, 2), 2, 4), 0, 0>>>(
          in_nnz, in_channel, in_feats.data_ptr<float>(), qkpos_ptr, in_csr_ptr,
          in_map_ptr, buffer.data_ptr<float>());
    }
  }

  // printf("in channel mod: %d", in_channel_mod);

  int buffer_offset = qsum_nnz * in_channel;

  if (data_type_half) {
    // naive_gemm_fp16_2<32, 4, 8>
    //     <<<dim3(DIV_UP(out_channel, 32), DIV_UP(qsum_nnz, 128), 1),
    //        dim3(16, 32, 1)>>>(
    //         kpos_ptr, qkpos_ptr, k_vol, in_channel, out_channel,
    //         reinterpret_cast<half *>(buffer.data_ptr<at::Half>()),
    //         reinterpret_cast<half *>(kernel.data_ptr<at::Half>()),
    //         reinterpret_cast<half *>(buffer.data_ptr<at::Half>()) +
    //             buffer_offset);
    wmma_ragged_gemm_fp16_wrapper(
        k_vol, qsum_nnz, buffer_offset,
        reinterpret_cast<half *>(kernel.data_ptr<at::Half>()),
        reinterpret_cast<half *>(buffer.data_ptr<at::Half>()), kpos_ptr,
        qkpos_ptr, in_channel, out_channel);
  } else {
    naive_gemm_fp32_2<32, 4, 8>
        <<<dim3(DIV_UP(out_channel, 32), DIV_UP(qsum_nnz, 128), 1),
           dim3(16, 32, 1)>>>(kpos_ptr, qkpos_ptr, k_vol, in_channel,
                              out_channel, buffer.data_ptr<float>(),
                              kernel.data_ptr<float>(),
                              buffer.data_ptr<float>() + buffer_offset);
  }

  /********************************************************************/
  // default stream

  if (data_type_half) {
    scatter_all_output_major_csr_half<<<
        DIV_UP(out_nnz, 4), dim3(DIV_UP(out_channel, 8), 4), 0, 0>>>(
        out_nnz, out_channel,
        (reinterpret_cast<half *>(buffer.data_ptr<at::Half>() + buffer_offset)),
        qkpos_ptr, out_csr_ptr, out_map_ptr,
        reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()));
  } else {
    scatter_all_output_major_csr_float<<<
        DIV_UP(out_nnz, 4), dim3(DIV_UP(out_channel, 4), 4), 0, 0>>>(
        out_nnz, out_channel, (buffer.data_ptr<float>() + buffer_offset),
        qkpos_ptr, out_csr_ptr, out_map_ptr, out_feats.data_ptr<float>());
  }

  if (separate_mid) {
    // computation for w[0, 0, 0]
    // in_nnz == out_nnz
    CUBLAS_CHECK(hipblasSetStream(cublasH, 0));
    if (data_type_half) {
      /*CUBLAS_CHECK(hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
              out_channel, in_nnz, in_channel,
              reinterpret_cast<half *>(alpha_half.data_ptr<at::Half>()),
              // &weight_ptr[mid_weight_id * in_channel * out_channel],
              reinterpret_cast<half *>(kernel.data_ptr<at::Half>()
                  + mid_weight_id * in_channel * out_channel),
              DataType, out_channel,
              reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()),
              DataType, in_channel,
              reinterpret_cast<half *>(alpha_half.data_ptr<at::Half>()),
              reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()),
              DataType, out_channel,
              ComputeType,
              CUBLAS_GEMM_DEFAULT_TENSOR_OP));*/

      CUBLAS_CHECK(hipblasHgemm(
          cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, out_channel, in_nnz, in_channel,
          reinterpret_cast<half *>(alpha_half.data_ptr<at::Half>()),
          // &weight_ptr[mid_weight_id * in_channel * out_channel],
          reinterpret_cast<half *>(kernel.data_ptr<at::Half>() +
                                   mid_weight_id * in_channel * out_channel),
          out_channel, reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()),
          in_channel,
          reinterpret_cast<half *>(alpha_copy_half.data_ptr<at::Half>()),
          reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()),
          out_channel));

      // at::addmm_out(out_feats, out_feats, in_feats, kernel[mid_weight_id]);

    } else {
      /*CUBLAS_CHECK(hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
              out_channel, in_nnz, in_channel,
              &alpha,
              // &weight_ptr[mid_weight_id * in_channel * out_channel],
              (kernel.data_ptr<float>() + mid_weight_id * in_channel *
         out_channel), DataType, out_channel, in_feats.data_ptr<float>(),
              DataType, in_channel,
              &alpha,
              out_feats.data_ptr<float>(),
              DataType, out_channel,
              ComputeType,
              CUBLAS_GEMM_DEFAULT_TENSOR_OP));*/

      CUBLAS_CHECK(hipblasSgemm(
          cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, out_channel, in_nnz, in_channel,
          &alpha,
          // &weight_ptr[mid_weight_id * in_channel * out_channel],
          (kernel.data_ptr<float>() + mid_weight_id * in_channel * out_channel),
          out_channel, in_feats.data_ptr<float>(), in_channel, &alpha_copy,
          out_feats.data_ptr<float>(), out_channel));

      // at::addmm_out(out_feats, out_feats, in_feats, kernel[mid_weight_id]);
    }
  }
}

void ConvolutionForwardBlockFused(
    const at::Tensor in_feats, const at::Tensor kernel, const int ksize_code,
    const int sum_nnz, at::Tensor out_feats, const at::Tensor kpos,
    const at::Tensor qkpos, const at::Tensor in_map, const at::Tensor out_map,
    const bool separate_mid, const bool TensorCoreFast) {

  int in_nnz = in_feats.size(0);
  int out_nnz = out_feats.size(0);
  int in_channel = in_feats.size(1);
  if (in_feats.size(1) != kernel.size(1)) {
    throw std::invalid_argument("Input feature size and kernel size mismatch");
  }
  int out_channel = kernel.size(2);
  int k_vol = kernel.size(0);

  bool data_type_half = in_feats.scalar_type() == at::ScalarType::Half;

  int *in_map_ptr = in_map.data_ptr<int>();
  int *out_map_ptr = out_map.data_ptr<int>();

  int ksx = ksize_code / 311;
  int ksy = (ksize_code - ksx * 311) / 17;
  int ksz = ksize_code - ksx * 311 - ksy * 17;
  int mid_weight_id =
      (ksx - 1) / 2 * ksy * ksz + (ksy - 1) / 2 * ksz + (ksz - 1) / 2;

  // cublas
  const float alpha = 1.0;
  const float beta = 0.0;
  at::Tensor alpha_half = torch::ones({1}, dtype(at::ScalarType::Half));
  at::Tensor beta_half = torch::zeros({1}, dtype(at::ScalarType::Half));

  hipblasComputeType_t ComputeType;
  hipDataType DataType;
  if (data_type_half) {
    ComputeType = HIPBLAS_COMPUTE_16F;
    DataType = HIP_R_16F;
  } else {
    ComputeType =
        TensorCoreFast ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
    DataType = HIP_R_32F;
  }

  hipblasHandle_t cublasH = at::cuda::getCurrentCUDABlasHandle();

  hipblasSetStream(cublasH, 0);

  hipblasSetMathMode(cublasH, HIPBLAS_TENSOR_OP_MATH);

  if (separate_mid) {
    if (data_type_half) {
      hipblasGemmEx(
          cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, out_channel, in_nnz, in_channel,
          reinterpret_cast<half *>(alpha_half.data_ptr<at::Half>()),
          // &weight_ptr[mid_weight_id * in_channel * out_channel],
          reinterpret_cast<half *>(kernel.data_ptr<at::Half>() +
                                   mid_weight_id * in_channel * out_channel),
          DataType, out_channel,
          // in_feats_ptr,
          reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()), DataType,
          in_channel, reinterpret_cast<half *>(beta_half.data_ptr<at::Half>()),
          // out_feats_ptr,
          reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()), DataType,
          out_channel, ComputeType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    } else {
      hipblasGemmEx(
          cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, out_channel, in_nnz, in_channel,
          &alpha,
          // &weight_ptr[mid_weight_id * in_channel * out_channel],
          (kernel.data_ptr<float>() + mid_weight_id * in_channel * out_channel),
          DataType, out_channel, in_feats.data_ptr<float>(), DataType,
          in_channel, &beta, out_feats.data_ptr<float>(), DataType, out_channel,
          ComputeType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    }
  }

  if (data_type_half) {
    /*if (in_channel % 8 == 0){
    fetch_on_demand_gemm_fp16_8<32, 4, 8><<<dim3(DIV_UP(out_channel, 32),
    DIV_UP(sum_nnz, 128), 1), dim3(4, 32, 1)>>>( kpos.data_ptr<int>(),
    qkpos.data_ptr<int>(), k_vol, in_channel, out_channel, reinterpret_cast<half
    *>(in_feats.data_ptr<at::Half>()), reinterpret_cast<half
    *>(kernel.data_ptr<at::Half>()), reinterpret_cast<half
    *>(out_feats.data_ptr<at::Half>()), in_map_ptr, out_map_ptr
    );
    fetch_on_demand_gemm_fp16_tc8_async<32, 4, 8, 16, 16, 16, 2>
        <<<dim3(DIV_UP(out_channel, 32), DIV_UP(sum_nnz, 128), 1), dim3(4, 32,
    1)>>>( kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel,
    out_channel, reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()),
            reinterpret_cast<half *>(kernel.data_ptr<at::Half>()),
            reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()),
            in_map_ptr, out_map_ptr
        );
    }*/
    if (in_channel % 4 == 0) {
      /*fetch_on_demand_gemm_fp16_4<32, 4, 8><<<dim3(DIV_UP(out_channel, 32),
      DIV_UP(sum_nnz, 128), 1), dim3(8, 32, 1)>>>( kpos.data_ptr<int>(),
      qkpos.data_ptr<int>(), k_vol, in_channel, out_channel,
          reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()),
          reinterpret_cast<half *>(kernel.data_ptr<at::Half>()),
          reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()),
          in_map_ptr, out_map_ptr
      );*/
      if (in_channel <= 16 || out_channel <= 16) {
        fetch_on_demand_gemm_fp16_4_once<16, 4, 8>
            <<<dim3(DIV_UP(out_channel, 16), DIV_UP(sum_nnz, 64), 1),
               dim3(4, 16, 1)>>>(
                kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel,
                out_channel,
                reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()),
                reinterpret_cast<half *>(kernel.data_ptr<at::Half>()),
                reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()),
                in_map_ptr, out_map_ptr);
      } else {
        fetch_on_demand_gemm_fp16_tc4_async<32, 4, 8, 16, 16, 16, 4, 2, 2>
            <<<dim3(DIV_UP(out_channel, 32), DIV_UP(sum_nnz, 128), 1),
               dim3(8, 32, 1)>>>(
                kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel,
                out_channel,
                reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()),
                reinterpret_cast<half *>(kernel.data_ptr<at::Half>()),
                reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()),
                in_map_ptr, out_map_ptr);
      }
    } else {
      fetch_on_demand_gemm_fp16_2<16, 8, 8>
          <<<dim3(DIV_UP(out_channel, 16), DIV_UP(sum_nnz, 128), 1),
             dim3(8, 16, 1)>>>(
              kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel,
              out_channel,
              reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()),
              reinterpret_cast<half *>(kernel.data_ptr<at::Half>()),
              reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()),
              in_map_ptr, out_map_ptr);
    }
  } else {
    if (in_channel % 4 == 0) {
      if (in_channel <= 16 && out_channel <= 16) {
        fetch_on_demand_gemm_fp32_once<16, 4, 8>
            <<<dim3(DIV_UP(out_channel, 16), DIV_UP(sum_nnz, 64), 1),
               dim3(4, 16, 1)>>>(
                kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel,
                out_channel, in_feats.data_ptr<float>(),
                kernel.data_ptr<float>(), out_feats.data_ptr<float>(),
                in_map_ptr, out_map_ptr);
      } else {
        fetch_on_demand_gemm_tf32<32, 4, 8, 16, 8, 16, 4, 2, 2>
            <<<dim3(DIV_UP(out_channel, 32), DIV_UP(sum_nnz, 128), 1),
               dim3(8, 32, 1)>>>(
                kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel,
                out_channel, in_feats.data_ptr<float>(),
                kernel.data_ptr<float>(), out_feats.data_ptr<float>(),
                in_map_ptr, out_map_ptr);
      }
    } else {
      gemm_float_fused_largeN_2<16, 8, 8>
          <<<dim3(DIV_UP(out_channel, 16), DIV_UP(sum_nnz, 128), 1),
             dim3(8, 16, 1)>>>(
              kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel,
              out_channel, in_feats.data_ptr<float>(), kernel.data_ptr<float>(),
              out_feats.data_ptr<float>(), in_map_ptr, out_map_ptr);
    }
  }
}

void ConvolutionBackward(const at::Tensor out_feats_grad,
                         const at::Tensor in_feats, const at::Tensor kernel,
                         const int ksize_code, const int sum_nnz,
                         at::Tensor in_feats_grad, at::Tensor kernel_grad,
                         const at::Tensor kernel_nnz,
                         const at::Tensor kernel_pos, const at::Tensor in_map,
                         const at::Tensor out_map, const at::Tensor in_csr,
                         const at::Tensor out_csr, at::Tensor buffer,
                         const bool TensorCoreMode) {

  int innz = in_feats.size(0);
  int onnz = out_feats_grad.size(0);
  bool separate_mid = (innz == onnz);
  int in_channel = in_feats.size(1);
  if (in_feats.size(1) != kernel.size(1)) {
    throw std::invalid_argument("Input feature size and kernel size mismatch");
  }
  int out_channel = kernel.size(2);
  int k_vol = kernel.size(0);

  float *ofeats_grad_ptr = out_feats_grad.data_ptr<float>();
  float *in_feats_ptr = in_feats.data_ptr<float>();
  float *weight_ptr = kernel.data_ptr<float>();

  float *ifeats_grad_ptr = in_feats_grad.data_ptr<float>();
  float *weight_grad_ptr = kernel_grad.data_ptr<float>();

  int *in_map_ptr = in_map.data_ptr<int>();
  int *out_map_ptr = out_map.data_ptr<int>();
  int *in_csr_ptr = in_csr.data_ptr<int>();
  int *out_csr_ptr = out_csr.data_ptr<int>();

  int *kpos_ptr = kernel_pos.data_ptr<int>();

  int ksx = ksize_code / 311;
  int ksy = (ksize_code - ksx * 311) / 17;
  int ksz = ksize_code - ksx * 311 - ksy * 17;
  int mid_weight_id =
      (ksx - 1) / 2 * ksy * ksz + (ksy - 1) / 2 * ksz + (ksz - 1) / 2;

  float *buf_ptr = buffer.data_ptr<float>();

  int buffer_offset = sum_nnz * in_channel;

  // cublas
  const float alpha = 1.0;
  const float beta = 0.0;

  hipblasHandle_t cublasH = at::cuda::getCurrentCUDABlasHandle();
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

  CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

  // gather for ofeats_grad
  gather_all_input_major_csr_float_4<<<
      DIV_UP(onnz, 4), dim3(DIV_UP(out_channel, 4), 2, 4), 0, 0>>>(
      onnz,            // in_nnz,
      out_channel,     // in_channel,
      ofeats_grad_ptr, // in_feats_ptr,
      kpos_ptr,
      out_csr_ptr,            // in_csr_ptr,
      out_map_ptr,            // in_map_ptr,
      &buf_ptr[buffer_offset] // buf_ptr
  );

  /*size_t const block_g = out_channel > PAR_THREAD ? out_channel : PAR_THREAD;
  size_t const grid_g = ((nnz) * (out_channel) + block_g - 1) / block_g;

  gather_all_input_major<<<grid_g, block_g>>>(
          nnz,
          k_vol,
          sum_nnz,
          kernel_pos_ptr,
          out_channel,
          ofeats_grad_ptr,
          out_map_ptr,
          out_buffer_ptr
  );*/

  // loop over all kernel offsets:
  // W^T X {\delta{out_feats}} = {\delta{in_feats}}^T
  int cur_idx = 0;

  for (int i = 0; i < k_vol; i++) {

    int cur_nnz = kernel_nnz.data_ptr<int>()[i];

    // TODO: put the zero check into the scheduler
    if (cur_nnz == 0) {
      continue;
    }

    // cublas GEMM for matmul
    if (TensorCoreMode) {
      CUBLAS_CHECK(hipblasGemmEx(
          cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, in_channel, cur_nnz, out_channel,
          &alpha, &weight_ptr[i * in_channel * out_channel], HIP_R_32F,
          out_channel, &buf_ptr[buffer_offset + cur_idx * out_channel],
          HIP_R_32F, out_channel, &beta, &buf_ptr[cur_idx * in_channel],
          HIP_R_32F, in_channel, HIPBLAS_COMPUTE_32F_FAST_16F,
          CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    } else {
      CUBLAS_CHECK(hipblasSgemm(
          cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, in_channel, cur_nnz, out_channel,
          &alpha, &weight_ptr[i * in_channel * out_channel], out_channel,
          &buf_ptr[buffer_offset + cur_idx * out_channel], out_channel, &beta,
          &buf_ptr[cur_idx * in_channel], in_channel));
    }
    cur_idx += cur_nnz;
  }

  // scatter for ifeats_grad

  scatter_all_output_major_csr_float<<<DIV_UP(innz, 4),
                                       dim3(DIV_UP(in_channel, 4), 4), 0, 0>>>(
      innz,       // out_nnz,
      in_channel, // out_channel,
      buf_ptr,    // &buf_ptr[buffer_offset],
      kpos_ptr,
      in_csr_ptr,     // out_csr_ptr,
      in_map_ptr,     // out_map_ptr,
      ifeats_grad_ptr // out_feats_ptr
  );

  /*size_t const block_s = in_channel > PAR_THREAD ? out_channel : PAR_THREAD;
  size_t const grid_s = (nnz * (in_channel) + block_s - 1) / block_s;

  scatter_all_output_major<<<grid_s, block_s>>>(
          nnz,
          k_vol,
          sum_nnz,
          kernel_pos_ptr,
          in_channel,
          in_buffer_ptr,
          in_map_ptr,
          ifeats_grad_ptr
  );*/

  // gather for in_feats
  gather_all_input_major_csr_float_4<<<
      DIV_UP(innz, 4), dim3(DIV_UP(in_channel, 4), 2, 4), 0, 0>>>(
      innz, in_channel, in_feats_ptr, kpos_ptr, in_csr_ptr, in_map_ptr,
      buf_ptr);
  /*gather_all_input_major<<<grid_s, block_s>>>(
          nnz,
          k_vol,
          sum_nnz,
          kernel_pos_ptr,
          in_channel,
          in_feats_ptr,
          in_map_ptr,
          in_buffer_ptr
  );*/

  // loop over all kernel offsets:
  // {\delta{out_feats}}^T X in_feats = {\delta{W}}^T
  // reset current idx in the map
  cur_idx = 0;

  for (int i = 0; i < k_vol; i++) {

    int cur_nnz = kernel_nnz.data_ptr<int>()[i];

    // TODO: put the zero check into the scheduler
    if (cur_nnz == 0) {
      continue;
    }

    // cublas GEMM for matmul
    if (TensorCoreMode) {
      CUBLAS_CHECK(hipblasGemmEx(
          cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, out_channel, in_channel, cur_nnz,
          &alpha, &buf_ptr[buffer_offset + cur_idx * out_channel], HIP_R_32F,
          out_channel, &buf_ptr[cur_idx * in_channel], HIP_R_32F, in_channel,
          &beta, &weight_grad_ptr[i * in_channel * out_channel], HIP_R_32F,
          out_channel, HIPBLAS_COMPUTE_32F_FAST_16F,
          CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    } else {
      CUBLAS_CHECK(hipblasSgemm(
          cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, out_channel, in_channel, cur_nnz,
          &alpha, &buf_ptr[buffer_offset + cur_idx * out_channel], out_channel,
          &buf_ptr[cur_idx * in_channel], in_channel, &beta,
          &weight_grad_ptr[i * in_channel * out_channel], out_channel));
    }
    cur_idx += cur_nnz;
  }

  // separate computation for center weight w[0, 0, 0]
  // computation for w[0, 0, 0]
  if (separate_mid) {
    if (TensorCoreMode) {

      CUBLAS_CHECK(hipblasSetMathMode(cublasH, HIPBLAS_TENSOR_OP_MATH));

      // W^T X {\delta{out_feats}} = {\delta{in_feats}}^T
      CUBLAS_CHECK(hipblasGemmEx(
          cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, in_channel, innz, out_channel,
          &alpha, &weight_ptr[mid_weight_id * in_channel * out_channel],
          HIP_R_32F, out_channel, ofeats_grad_ptr, HIP_R_32F, out_channel,
          &alpha, ifeats_grad_ptr, HIP_R_32F, in_channel,
          HIPBLAS_COMPUTE_32F_FAST_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));

      // {\delta{out_feats}}^T X in_feats = {\delta{W}}^T
      CUBLAS_CHECK(hipblasGemmEx(
          cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, out_channel, in_channel, innz,
          &alpha, ofeats_grad_ptr, HIP_R_32F, out_channel, in_feats_ptr,
          HIP_R_32F, in_channel, &alpha,
          &weight_grad_ptr[mid_weight_id * in_channel * out_channel],
          HIP_R_32F, out_channel, HIPBLAS_COMPUTE_32F_FAST_16F,
          CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    } else {

      // W^T X {\delta{out_feats}} = {\delta{in_feats}}^T
      CUBLAS_CHECK(hipblasSgemm(
          cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, in_channel, innz, out_channel,
          &alpha, &weight_ptr[mid_weight_id * in_channel * out_channel],
          out_channel, ofeats_grad_ptr, out_channel, &alpha, ifeats_grad_ptr,
          in_channel));

      // {\delta{out_feats}}^T X in_feats = {\delta{W}}^T
      CUBLAS_CHECK(hipblasSgemm(
          cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, out_channel, in_channel, innz,
          &alpha, ofeats_grad_ptr, out_channel, in_feats_ptr, in_channel,
          &alpha, &weight_grad_ptr[mid_weight_id * in_channel * out_channel],
          out_channel));
    }
  }

  return;
}
