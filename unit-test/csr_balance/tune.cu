#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define _FLOAT4(pointer) (reinterpret_cast<float4 *>(&(pointer))[0])

inline __device__ int kernel_decoder(int code){
    return (code / 1186111);
}

inline __device__ int kernel_map_decoder(int code){
    return (code % 1186111);
}

__device__ __forceinline__ int binary_search_find_nnz(
                            const int *S_csrRowPtr, const int eid, 
                            const int start, const int end) {
    
    int lo = start, hi = end;
    if (lo == hi){
        return lo;
    }
    while (lo < hi) {
        int mid = (lo + hi) >> 1;
        if (__ldg(S_csrRowPtr + mid) <= eid) {
            lo = mid + 1;
        } else {
            hi = mid;
        }
    }
    if (__ldg(S_csrRowPtr + hi) <= eid) {
        return hi;
    } else {
        return hi - 1;
    }
}


__global__ void gather_all_input_major_csr_template(
                    const int nnz, 
                    const int kv, 
                    const int total_knnz, 
                    const int *__restrict__ knnz_pos, 
                    const int c_in, 
                    const float *__restrict__ in_f, 
                    const int *__restrict__ icsr, 
                    const int *__restrict__ imap, 
                    float *g_f){
    // id-th nnz
    const int id = blockIdx.x * block_size_z + threadIdx.z;  
    if (id >= nnz){return;}
    const int m_start = __ldg(&icsr[id]);
    const int m_end = __ldg(&icsr[id + 1]);
#pragma unroll
    for (int k = m_start; ; k += block_size_y){
        int kp = k + threadIdx.y;
        // make sure  m_start <= kp < m_end
        if (kp >= m_end){break;}
        int kinf = __ldg(&imap[kp]);
        // which kernel offset
        int kofs = kernel_decoder(kinf);
        int buf_ofs = kernel_map_decoder(kinf);
        int buf_start = __ldg(&knnz_pos[kofs]);
        int buf_pos = buf_start + buf_ofs;
#pragma unroll
        for (int c = 0; ; c += block_size_x){
            // which input channel
            int cp = (c + threadIdx.x) << 2;
            if (cp >= c_in){break;}
            (g_f[buf_pos * c_in + cp]) = 
                __ldg(&in_f[id * c_in + cp]);
        }
    }
}


__global__ void gather_all_input_major_csr_balance(
                    const int nnz, 
                    const int kv, 
                    const int total_knnz, 
                    const int *__restrict__ knnz_pos, 
                    const int c_in, 
                    float *in_f, 
                    const int *__restrict__ icsr, 
                    const int *__restrict__ imap, 
                    float *g_f){
    // [m_start, m_end]-th mapping
    const int m_start = blockIdx.x * _MPNS_PER_BLOCK;  
    const int m_end = min(m_start + _MPNS_PER_BLOCK, total_knnz) - 1;
    // store the starting id of the block
    __shared__ int nid[block_size_y];
    nid[threadIdx.y] = binary_search_find_nnz(icsr, m_start, 0, nnz);
    // bnid = binary_search_find_nnz(icsr, m_start, 0, nnz);
    // register to store the specific id of the thread
    // int nid = binary_search_find_nnz(icsr, m_start, 0, nnz);
#pragma unroll
    for (int k = m_start; ; k += block_size_y){
        int kp = k + threadIdx.y;
        // make sure  m_start <= kp <= m_end
        if (kp > m_end){break;}
         // which nnz
        while (kp >= icsr[nid[threadIdx.y] + 1]){
            nid[threadIdx.y] += 1;}
        // which kernel offset
        int kinf = __ldg(&imap[kp]);
        int kofs = kernel_decoder(kinf);
        int buf_ofs = kernel_map_decoder(kinf);
        int buf_start = __ldg(&knnz_pos[kofs]);
        int buf_pos = buf_start + buf_ofs;
#pragma unroll
        for (int c = 0; ; c += block_size_x){
            // which input channel
            int cp = (c + threadIdx.x) << 2;
            if (cp >= c_in){break;}
            _FLOAT4(g_f[buf_pos * c_in + cp]) = 
                _FLOAT4(in_f[nid[threadIdx.y] * c_in + cp]);
        }    
    }
}