#include "hip/hip_runtime.h"
#include "spconv.h"
#include "spconv.cuh"

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <torch/extension.h>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "cublas_utils.h"

#define DIV_UP(x, y) (x + y - 1) / y

extern "C"

void ConvolutionForward(at::Tensor in_feats, 
                        at::Tensor kernel, 
                        const int ksize_code, 
                        const int sum_nnz, 
                        at::Tensor out_feats, 
                        const at::Tensor kernel_nnz, 
                        const at::Tensor kernel_pos, 
                        const at::Tensor in_map, 
                        const at::Tensor out_map, 
                        const at::Tensor in_csr, 
                        const at::Tensor out_csr, 
                        at::Tensor buffer, 
                        const bool separate_mid, 
                        const bool TensorCore16Fast
                        ){
    
    // printf("[SubmanifoldSparseConv] - Starts.\n");

    int in_nnz = in_feats.size(0);
    int out_nnz = out_feats.size(0);
    int in_channel = in_feats.size(1);
    int in_channel_mod = in_channel;
    if (in_feats.size(1) != kernel.size(1)) {
        throw std::invalid_argument("Input feature size and kernel size mismatch");
    }
    int out_channel = kernel.size(2);
    int k_vol = kernel.size(0);

    bool data_type_half = in_feats.scalar_type() == at::ScalarType::Half;
   
    int *in_map_ptr = in_map.data_ptr<int>();
    int *out_map_ptr = out_map.data_ptr<int>();
    int *in_csr_ptr = in_csr.data_ptr<int>();
    int *out_csr_ptr = out_csr.data_ptr<int>();
    int *kpos_ptr = kernel_pos.data_ptr<int>();

    // int sum_nnz = in_buffer.size(0);
    // printf("sum nnz: %d", sum_nnz);

    int ksx = ksize_code / 94273;
    int ksy = (ksize_code - ksx * 94273) / 311;
    int ksz = ksize_code - ksx * 94273 - ksy * 311;
    int mid_weight_id = (ksx - 1) / 2 * ksy * ksz + 
        (ksy - 1) / 2 * ksz + (ksz - 1) / 2;

    // cublas
    const float alpha = 1.0;
    const float alpha_copy = 1.0;
    const float beta = 0.0;
    at::Tensor alpha_half = torch::ones({1}, dtype(at::ScalarType::Half));
    at::Tensor alpha_copy_half = torch::ones({1}, dtype(at::ScalarType::Half));
    at::Tensor beta_half = torch::zeros({1}, dtype(at::ScalarType::Half));

    hipblasHandle_t cublasH = at::cuda::getCurrentCUDABlasHandle();

    CUBLAS_CHECK(hipblasSetMathMode(cublasH, HIPBLAS_TENSOR_OP_MATH));

    hipblasComputeType_t ComputeType;
    hipDataType DataType;
    if (data_type_half){
        ComputeType = HIPBLAS_COMPUTE_16F;
        DataType = HIP_R_16F;
    }
    else{
        ComputeType = TensorCore16Fast ? 
            HIPBLAS_COMPUTE_32F_FAST_16F : HIPBLAS_COMPUTE_32F_FAST_TF32;
        DataType = HIP_R_32F;
    }

    /********************************************************************/
    // default stream

    if (data_type_half){

        gather_all_input_major_csr_half_4<<<DIV_UP(in_nnz, 4), 
                dim3(DIV_UP(in_channel, 4), 2, 4), 0, 0>>>(
                    in_nnz, in_channel, reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()), 
                    kpos_ptr, in_csr_ptr, in_map_ptr, reinterpret_cast<half *>(buffer.data_ptr<at::Half>())
        );

    }
    else{

        gather_all_input_major_csr_float_4<<<DIV_UP(in_nnz, 4), 
                dim3(DIV_UP(in_channel, 4), 2, 4), 0, 0>>>(
                    in_nnz, in_channel, in_feats.data_ptr<float>(), kpos_ptr, 
                    in_csr_ptr, in_map_ptr, buffer.data_ptr<float>()
        );
         
    }

    // printf("in channel mod: %d", in_channel_mod);

    int buffer_offset = sum_nnz * in_channel;

    /********************************************************************/
    // create the streams
    int n_stream = 4;

    hipStream_t *pl_stream;
    pl_stream = (hipStream_t *)new hipStream_t[n_stream];
    
    for (int i = 0; i < n_stream; i++) {
        hipStreamCreateWithFlags(&pl_stream[i], hipStreamDefault);
    }

    /********************************************************************/
    // loop over all kernel offsets
    int cur_idx = 0;

    // printf("The GemmEx is used here.\n");
    // Suppose an odd kernel size
    for (int i = 0; i < k_vol; i++){

        int cur_nnz = kernel_nnz.data_ptr<int>()[i];
        
        // TODO: put the zero check into the scheduler
        if (cur_nnz == 0){continue;}

        int stream_id = i % n_stream;

        CUBLAS_CHECK(hipblasSetStream(cublasH, pl_stream[stream_id]));

        if (data_type_half){
            // cublas GEMM for matmul
            /*CUBLAS_CHECK(hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    out_channel, cur_nnz, in_channel, 
                    (reinterpret_cast<half *>(alpha_half.data_ptr<at::Half>())),
                    // &weight_ptr[i * in_channel * out_channel], 
                    (reinterpret_cast<half *>(kernel.data_ptr<at::Half>()
                        + i * in_channel * out_channel)),
                    DataType, out_channel, 
                    // &buf_ptr[cur_idx * in_channel], 
                    (reinterpret_cast<half *>(buffer.data_ptr<at::Half>()
                        + cur_idx * in_channel)),
                    DataType, in_channel, 
                    (reinterpret_cast<half *>(beta_half.data_ptr<at::Half>())),  
                    // &buf_ptr[buffer_offset + cur_idx * out_channel], 
                    (reinterpret_cast<half *>(buffer.data_ptr<at::Half>()
                        + buffer_offset + cur_idx * out_channel)), 
                    DataType, out_channel,
                    ComputeType,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP));*/
            
            CUBLAS_CHECK(hipblasHgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    out_channel, cur_nnz, in_channel, 
                    (reinterpret_cast<half *>(alpha_half.data_ptr<at::Half>())), 
                    // &weight_ptr[i * in_channel * out_channel], 
                    (reinterpret_cast<half *>(kernel.data_ptr<at::Half>()
                        + i * in_channel * out_channel)),
                    out_channel, 
                    // &buf_ptr[cur_idx * in_channel], 
                    (reinterpret_cast<half *>(buffer.data_ptr<at::Half>()
                        + cur_idx * in_channel)),
                    in_channel, 
                    (reinterpret_cast<half *>(beta_half.data_ptr<at::Half>())), 
                    // &buf_ptr[buffer_offset + cur_idx * out_channel], 
                    (reinterpret_cast<half *>(buffer.data_ptr<at::Half>()
                        + buffer_offset + cur_idx * out_channel)), 
                    out_channel));
        }
        else{
            // cublas GEMM for matmul
            /*CUBLAS_CHECK(hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    out_channel, cur_nnz, in_channel, 
                    &alpha, 
                    // &weight_ptr[i * in_channel * out_channel], 
                    (kernel.data_ptr<float>() + i * in_channel * out_channel),
                    DataType, out_channel, 
                    // &buf_ptr[cur_idx * in_channel], 
                    (buffer.data_ptr<float>() + cur_idx * in_channel),
                    DataType, in_channel, 
                    &beta, 
                    // &buf_ptr[buffer_offset + cur_idx * out_channel], 
                    (buffer.data_ptr<float>() + buffer_offset + cur_idx * out_channel), 
                    DataType, out_channel,
                    ComputeType,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP));*/
            
            CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    out_channel, cur_nnz, in_channel, 
                    &alpha, 
                    // &weight_ptr[i * in_channel * out_channel], 
                    (kernel.data_ptr<float>() + i * in_channel * out_channel),
                    out_channel, 
                    // &buf_ptr[cur_idx * in_channel], 
                    (buffer.data_ptr<float>() + cur_idx * in_channel),
                    in_channel, 
                    &beta, 
                    // &buf_ptr[buffer_offset + cur_idx * out_channel], 
                    (buffer.data_ptr<float>() + buffer_offset + cur_idx * out_channel), 
                    out_channel));
        }

        cur_idx += cur_nnz;
    }

    hipDeviceSynchronize();
    for (int i = 0; i < n_stream; i++) {
        hipStreamDestroy(pl_stream[i]);
    } 

    /********************************************************************/
    // default stream

    if (data_type_half){
        scatter_all_output_major_csr_half<<<DIV_UP(out_nnz, 4), 
            dim3(DIV_UP(out_channel, 8), 4), 0, 0>>>(
                out_nnz, out_channel, (reinterpret_cast<half *>(buffer.data_ptr<at::Half>() + buffer_offset)), 
                kpos_ptr, out_csr_ptr, out_map_ptr, reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()));
    }
    else{
        scatter_all_output_major_csr_float<<<DIV_UP(out_nnz, 4), 
            dim3(DIV_UP(out_channel, 4), 4), 0, 0>>>(
                out_nnz, out_channel, (buffer.data_ptr<float>() + buffer_offset), kpos_ptr, 
                out_csr_ptr, out_map_ptr, out_feats.data_ptr<float>());
    }

    if (separate_mid){
    // computation for w[0, 0, 0]
    // in_nnz == out_nnz
        CUBLAS_CHECK(hipblasSetStream(cublasH, 0));
        if (data_type_half){
            /*CUBLAS_CHECK(hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    out_channel, in_nnz, in_channel, 
                    reinterpret_cast<half *>(alpha_half.data_ptr<at::Half>()), 
                    // &weight_ptr[mid_weight_id * in_channel * out_channel], 
                    reinterpret_cast<half *>(kernel.data_ptr<at::Half>() 
                        + mid_weight_id * in_channel * out_channel),
                    DataType, out_channel, 
                    reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()), 
                    DataType, in_channel, 
                    reinterpret_cast<half *>(alpha_half.data_ptr<at::Half>()), 
                    reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()), 
                    DataType, out_channel,
                    ComputeType,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP));*/
            
            CUBLAS_CHECK(hipblasHgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    out_channel, in_nnz, in_channel, 
                    reinterpret_cast<half *>(alpha_half.data_ptr<at::Half>()), 
                    // &weight_ptr[mid_weight_id * in_channel * out_channel], 
                    reinterpret_cast<half *>(kernel.data_ptr<at::Half>() 
                        + mid_weight_id * in_channel * out_channel),
                    out_channel, 
                    reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()), 
                    in_channel, 
                    reinterpret_cast<half *>(alpha_copy_half.data_ptr<at::Half>()), 
                    reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()), 
                    out_channel));

        }
        else{
            /*CUBLAS_CHECK(hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    out_channel, in_nnz, in_channel, 
                    &alpha, 
                    // &weight_ptr[mid_weight_id * in_channel * out_channel], 
                    (kernel.data_ptr<float>() + mid_weight_id * in_channel * out_channel),
                    DataType, out_channel, 
                    in_feats.data_ptr<float>(), 
                    DataType, in_channel, 
                    &alpha, 
                    out_feats.data_ptr<float>(), 
                    DataType, out_channel,
                    ComputeType,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP));*/
                    
            CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    out_channel, in_nnz, in_channel, 
                    &alpha, 
                    // &weight_ptr[mid_weight_id * in_channel * out_channel], 
                    (kernel.data_ptr<float>() + mid_weight_id * in_channel * out_channel),
                    out_channel, 
                    in_feats.data_ptr<float>(), 
                    in_channel, 
                    &alpha_copy, 
                    out_feats.data_ptr<float>(), 
                    out_channel));
        }
    }
}


void ConvolutionForward_simple(
                        const at::Tensor in_feats, 
                        const at::Tensor kernel, 
                        const int ksize_code, 
                        const int sum_nnz, 
                        at::Tensor out_feats, 
                        const at::Tensor kpos, 
                        const at::Tensor qkpos, 
                        const at::Tensor in_map, 
                        const at::Tensor out_map, 
                        const bool separate_mid, 
                        const bool TensorCoreFast
                        ){
    
    // printf("[SubmanifoldSparseConv] - Starts.\n");

    int in_nnz = in_feats.size(0);
    int out_nnz = out_feats.size(0);
    int in_channel = in_feats.size(1);
    if (in_feats.size(1) != kernel.size(1)) {
        throw std::invalid_argument("Input feature size and kernel size mismatch");
    }
    int out_channel = kernel.size(2);
    int k_vol = kernel.size(0);

    bool data_type_half = in_feats.scalar_type() == at::ScalarType::Half;

    int *in_map_ptr = in_map.data_ptr<int>();
    int *out_map_ptr = out_map.data_ptr<int>();
    
    int ksx = ksize_code / 94273;
    int ksy = (ksize_code - ksx * 94273) / 311;
    int ksz = ksize_code - ksx * 94273 - ksy * 311;
    int mid_weight_id = (ksx - 1) / 2 * ksy * ksz + 
        (ksy - 1) / 2 * ksz + (ksz - 1) / 2;

    // cublas
    const float alpha = 1.0;
    const float beta = 0.0;
    at::Tensor alpha_half = torch::ones({1}, dtype(at::ScalarType::Half));
    at::Tensor beta_half = torch::zeros({1}, dtype(at::ScalarType::Half));

    hipblasComputeType_t ComputeType;
    hipDataType DataType;
    if (data_type_half){
        ComputeType = HIPBLAS_COMPUTE_16F;
        DataType = HIP_R_16F;
    }
    else{
        ComputeType = TensorCoreFast ? 
            HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
        DataType = HIP_R_32F;
    }

    hipblasHandle_t cublasH = at::cuda::getCurrentCUDABlasHandle();

    hipblasSetStream(cublasH, 0);

    hipblasSetMathMode(cublasH, HIPBLAS_TENSOR_OP_MATH);

    if (separate_mid){
        if (data_type_half){
            hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    out_channel, in_nnz, in_channel, 
                    reinterpret_cast<half *>(alpha_half.data_ptr<at::Half>()), 
                    // &weight_ptr[mid_weight_id * in_channel * out_channel], 
                    reinterpret_cast<half *>(kernel.data_ptr<at::Half>() 
                        + mid_weight_id * in_channel * out_channel),
                    DataType, out_channel, 
                    // in_feats_ptr, 
                    reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()), 
                    DataType, in_channel, 
                    reinterpret_cast<half *>(beta_half.data_ptr<at::Half>()), 
                    // out_feats_ptr, 
                    reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()), 
                    DataType, out_channel,
                    ComputeType,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
        else{
            hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    out_channel, in_nnz, in_channel, 
                    &alpha, 
                    // &weight_ptr[mid_weight_id * in_channel * out_channel], 
                    (kernel.data_ptr<float>() + mid_weight_id * in_channel * out_channel),
                    DataType, out_channel, 
                    in_feats.data_ptr<float>(), 
                    DataType, in_channel, 
                    &beta, 
                    out_feats.data_ptr<float>(), 
                    DataType, out_channel,
                    ComputeType,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
    }

    if (data_type_half){
        if (in_channel % 4 == 0){  
            fused_gemm_fp16_c4<16, 4, 8>
                    <<<dim3(DIV_UP(out_channel, 16), DIV_UP(sum_nnz, 64), 1), dim3(4, 16, 1)>>>(
                    kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel, out_channel, 
                    reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()),
                    reinterpret_cast<half *>(kernel.data_ptr<at::Half>()),
                    reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()),
                    in_map_ptr, out_map_ptr
                );
            }
        else{
            fused_gemm_fp16_c2<16, 4, 8>
                    <<<dim3(DIV_UP(out_channel, 16), DIV_UP(sum_nnz, 64), 1), dim3(8, 16, 1)>>>(
                    kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel, out_channel, 
                    reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()),
                    reinterpret_cast<half *>(kernel.data_ptr<at::Half>()),
                    reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()),
                    in_map_ptr, out_map_ptr
                );
        }
    }
    else{
        if(in_channel % 4 == 0){
            fused_gemm_fp32_c4<16, 4, 8>
                    <<<dim3(DIV_UP(out_channel, 16), DIV_UP(sum_nnz, 64), 1), dim3(4, 16, 1)>>>(
                    kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel, out_channel, 
                    in_feats.data_ptr<float>(), kernel.data_ptr<float>(), out_feats.data_ptr<float>(), 
                    in_map_ptr, out_map_ptr
                );
        }
        else{
            fused_gemm_fp32_c2<16, 4, 8>
                    <<<dim3(DIV_UP(out_channel, 16), DIV_UP(sum_nnz, 64), 1), dim3(8, 16, 1)>>>(
                    kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel, out_channel, 
                    in_feats.data_ptr<float>(), kernel.data_ptr<float>(), out_feats.data_ptr<float>(), 
                    in_map_ptr, out_map_ptr
            );
        }
    }
}


void ConvolutionForward_naive(
                        const at::Tensor in_feats, 
                        const at::Tensor kernel, 
                        const int ksize_code, 
                        const int sum_nnz, 
                        at::Tensor out_feats, 
                        const at::Tensor kernel_nnz, 
                        const at::Tensor kernel_pos, 
                        const at::Tensor in_map, 
                        const at::Tensor out_map, 
                        const bool separate_mid, 
                        const bool TensorCoreFast
                        ){
    
    int in_nnz = in_feats.size(0);
    int out_nnz = out_feats.size(0);
    int in_channel = in_feats.size(1);
    if (in_feats.size(1) != kernel.size(1)) {
        throw std::invalid_argument("Input feature size and kernel size mismatch");
    }
    int out_channel = kernel.size(2);
    int k_vol = kernel.size(0);

    bool data_type_half = in_feats.scalar_type() == at::ScalarType::Half;

    int *in_map_ptr = in_map.data_ptr<int>();
    int *out_map_ptr = out_map.data_ptr<int>();
    int *kpos_ptr = kernel_pos.data_ptr<int>();

    // int sum_nnz = in_buffer.size(0);
    // int buffer_offset = sum_nnz * in_channel;
    // printf("sum nnz: %d", sum_nnz);

    int ksx = ksize_code / 94273;
    int ksy = (ksize_code - ksx * 94273) / 311;
    int ksz = ksize_code - ksx * 94273 - ksy * 311;
    int mid_weight_id = (ksx - 1) / 2 * ksy * ksz + 
        (ksy - 1) / 2 * ksz + (ksz - 1) / 2;

    // float *buf_ptr = buffer.data_ptr<float>();

    // cublas
    const float alpha = 1.0;
    const float beta = 0.0;
    at::Tensor alpha_half = torch::ones({1}, dtype(at::ScalarType::Half));
    at::Tensor beta_half = torch::zeros({1}, dtype(at::ScalarType::Half));

    hipblasComputeType_t ComputeType;
    hipDataType DataType;
    if (data_type_half){
        ComputeType = HIPBLAS_COMPUTE_16F;
        DataType = HIP_R_16F;
    }
    else{
        ComputeType = TensorCoreFast ? 
            HIPBLAS_COMPUTE_32F_FAST_16F : HIPBLAS_COMPUTE_32F_FAST_TF32;
        DataType = HIP_R_32F;
    }

    hipblasHandle_t cublasH = at::cuda::getCurrentCUDABlasHandle();

    CUBLAS_CHECK(hipblasSetStream(cublasH, 0));

    CUBLAS_CHECK(hipblasSetMathMode(cublasH, HIPBLAS_TENSOR_OP_MATH));

    if (separate_mid){
    // computation for w[0, 0, 0]
    // in_nnz == out_nnz
        if (data_type_half){
            CUBLAS_CHECK(hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    out_channel, in_nnz, in_channel, 
                    reinterpret_cast<half *>(alpha_half.data_ptr<at::Half>()), 
                    // &weight_ptr[mid_weight_id * in_channel * out_channel], 
                    reinterpret_cast<half *>(kernel.data_ptr<at::Half>() 
                        + mid_weight_id * in_channel * out_channel),
                    DataType, out_channel, 
                    // in_feats_ptr, 
                    reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()), 
                    DataType, in_channel, 
                    reinterpret_cast<half *>(beta_half.data_ptr<at::Half>()), 
                    // out_feats_ptr, 
                    reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()), 
                    DataType, out_channel,
                    ComputeType,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP));
        }
        else{
            CUBLAS_CHECK(hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    out_channel, in_nnz, in_channel, 
                    &alpha, 
                    // &weight_ptr[mid_weight_id * in_channel * out_channel], 
                    (kernel.data_ptr<float>() + mid_weight_id * in_channel * out_channel),
                    DataType, out_channel, 
                    in_feats.data_ptr<float>(), 
                    DataType, in_channel, 
                    &beta, 
                    out_feats.data_ptr<float>(), 
                    DataType, out_channel,
                    ComputeType,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP));
        }
    }

    /********************************************************************/
    // create the streams
    /*int n_stream = 4;

    hipStream_t *pl_stream;
    pl_stream = (hipStream_t *)new hipStream_t[n_stream];
    
    for (int i = 0; i < n_stream; i++) {
        hipStreamCreateWithFlags(&pl_stream[i], hipStreamDefault);
    }*/

    /********************************************************************/
    // loop over all kernel offsets
    int cur_idx = 0;
    // int stream_id = 0;
    while(cur_idx < k_vol){
      
        // decide the grid dim
        size_t gridnum_x = DIV_UP(out_channel, 16);
        size_t gridnum_y = 0;

        // decide how many gemm to be fused
        int fusion_kofs = 0;
        int all_count = 0;
        int cur_count = 0;
        while (all_count < 16000){
            cur_count = kernel_nnz.data_ptr<int>()[cur_idx + fusion_kofs];
            all_count += cur_count;
            gridnum_y += DIV_UP(cur_count, 16);
            fusion_kofs += 1;
            if (cur_idx + fusion_kofs >= k_vol){break;}
        }

        // printf("-----\n");
        // printf("cur_idx: %d, all count: %d, fusion_kofs: %d\n", cur_idx, all_count, fusion_kofs);
        if (gridnum_y == 0){cur_idx += fusion_kofs; continue;}
       
        /*// call horizontal_fused_gemm
        horizontal_fused_gemm<16><<<dim3(gridnum_x, gridnum_y, fusion_kofs), dim3(16, 16, 1)>>>(
            max_count, fusion_kofs, in_channel, out_channel, &kpos_ptr[cur_idx], in_feats_ptr, 
            &weight_ptr[weight_id * in_channel * out_channel], out_feats_ptr, in_map_ptr, out_map_ptr, skip_idx
        );*/
        if (data_type_half){
            block_fused_gemm_half<16><<<dim3(gridnum_x, gridnum_y, 1), dim3(16, 16, 1)>>>(
                fusion_kofs, in_channel, out_channel, &kpos_ptr[cur_idx], 
                reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()), 
                reinterpret_cast<half *>(kernel.data_ptr<at::Half>() + cur_idx * in_channel * out_channel),
                reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()), in_map_ptr, out_map_ptr
            );
        }
        else{
            block_fused_gemm_float<16><<<dim3(gridnum_x, gridnum_y, 1), dim3(16, 16, 1)>>>(
                fusion_kofs, in_channel, out_channel, &kpos_ptr[cur_idx], 
                in_feats.data_ptr<float>(), 
                (kernel.data_ptr<float>() + cur_idx * in_channel * out_channel),
                out_feats.data_ptr<float>(), in_map_ptr, out_map_ptr
            );
        }

      cur_idx += fusion_kofs;
      // stream_id = (stream_id + 1) % 4;
    }

    /*hipDeviceSynchronize();
    for (int i = 0; i < n_stream; i++) {
        hipStreamDestroy(pl_stream[i]);
    }*/
}


/*
Only FP32 available for ablation study purpose.
*/
void ConvolutionForward_batched(
                        const at::Tensor in_feats, 
                        const at::Tensor kernel, 
                        const int ksize_code, 
                        const int sum_nnz, 
                        at::Tensor out_feats, 
                        const at::Tensor kernel_nnz, 
                        const at::Tensor kernel_pos, 
                        const at::Tensor in_map, 
                        const at::Tensor out_map, 
                        const bool separate_mid, 
                        const int M,
                        const float theta
                        ){

    int in_nnz = in_feats.size(0);
    int out_nnz = out_feats.size(0);
    int in_channel = in_feats.size(1);
    if (in_feats.size(1) != kernel.size(1)) {
        throw std::invalid_argument("Input feature size and kernel size mismatch");
    }
    int out_channel = kernel.size(2);
    int k_vol = kernel.size(0);

    float *in_feats_ptr = in_feats.data_ptr<float>();
    float *weight_ptr = kernel.data_ptr<float>();
    float *out_feats_ptr = out_feats.data_ptr<float>();
    int *in_map_ptr = in_map.data_ptr<int>();
    int *out_map_ptr = out_map.data_ptr<int>();

    int *kpos_ptr = kernel_pos.data_ptr<int>();

    int ksx = ksize_code / 94273;
    int ksy = (ksize_code - ksx * 94273) / 311;
    int ksz = ksize_code - ksx * 94273 - ksy * 311;
    int mid_weight_id = (ksx - 1) / 2 * ksy * ksz + 
        (ksy - 1) / 2 * ksz + (ksz - 1) / 2;

    // cublas
    const float alpha = 1.0;
    const float beta = 0.0;

    hipblasHandle_t cublasH = at::cuda::getCurrentCUDABlasHandle();

    CUBLAS_CHECK(hipblasSetStream(cublasH, 0));

    if (separate_mid){
    
        CUBLAS_CHECK(hipblasSetMathMode(cublasH, HIPBLAS_TENSOR_OP_MATH));
        
        CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    out_channel, in_nnz, in_channel, 
                    &alpha, 
                    &weight_ptr[mid_weight_id * in_channel * out_channel], 
                    out_channel, 
                    in_feats_ptr, 
                    in_channel, 
                    &beta, 
                    out_feats_ptr, 
                    out_channel));

    }

    int cur_k = 0;
    // int stream_id = 0;
    while(cur_k < k_vol){

        // decide how many gemms to be batched
        int fusion_kofs = 0;
        int max_count = 0;
        int bound_count = 0;
        int min_count = in_nnz;
        int cur_count = 0;
        int all_count = 0;
        while ((cur_k + fusion_kofs < k_vol) && (all_count < M)){
            cur_count = kernel_nnz.data_ptr<int>()[cur_k + fusion_kofs];
            if (cur_count == 0) {fusion_kofs += 1; continue;}
            max_count = max(cur_count, max_count);
            min_count = min(cur_count, min_count);
            if (1 - (double)min_count / (double)max_count > theta){break;}
            fusion_kofs += 1;
            all_count += cur_count;
            bound_count = max_count;
        }

        // printf("-----\n");
        // printf("cur_k: %d, min count: %d, max count: %d, fusion_kofs: %d\n", cur_k, min_count, max_count, fusion_kofs);

        // decide the grid dim
        size_t gridnum_x = DIV_UP(out_channel, 16);
        size_t gridnum_y = DIV_UP(bound_count, 64);
       
        // call batched_gemm
        batched_gemm_fp32_c4<16, 4, 8><<<dim3(gridnum_x, gridnum_y, fusion_kofs), dim3(4, 16, 1)>>>(
            &kpos_ptr[cur_k], k_vol, in_channel, out_channel, 
            in_feats.data_ptr<float>(), &weight_ptr[cur_k * in_channel * out_channel], out_feats.data_ptr<float>(), 
            in_map_ptr, out_map_ptr
        );

      cur_k += fusion_kofs;
      // stream_id = (stream_id + 1) % 4;
    }
}


void ConvolutionForward_separate(
                        const at::Tensor in_feats, 
                        const at::Tensor kernel, 
                        const int ksize_code, 
                        const int sum_nnz, 
                        at::Tensor out_feats, 
                        const at::Tensor kernel_nnz, 
                        const at::Tensor kernel_pos, 
                        const at::Tensor in_map, 
                        const at::Tensor out_map, 
                        const bool separate_mid
                        ){
                        
    int in_nnz = in_feats.size(0);
    int out_nnz = out_feats.size(0);
    int in_channel = in_feats.size(1);
    if (in_feats.size(1) != kernel.size(1)) {
        throw std::invalid_argument("Input feature size and kernel size mismatch");
    }
    int out_channel = kernel.size(2);
    int k_vol = kernel.size(0);

    float *in_feats_ptr = in_feats.data_ptr<float>();
    float *weight_ptr = kernel.data_ptr<float>();
    float *out_feats_ptr = out_feats.data_ptr<float>();
    int *in_map_ptr = in_map.data_ptr<int>();
    int *out_map_ptr = out_map.data_ptr<int>();

    int *kpos_ptr = kernel_pos.data_ptr<int>();

    // int sum_nnz = in_buffer.size(0);
    // int buffer_offset = sum_nnz * in_channel;
    // printf("sum nnz: %d", sum_nnz);

    int ksx = ksize_code / 94273;
    int ksy = (ksize_code - ksx * 94273) / 311;
    int ksz = ksize_code - ksx * 94273 - ksy * 311;
    int mid_weight_id = (ksx - 1) / 2 * ksy * ksz + 
        (ksy - 1) / 2 * ksz + (ksz - 1) / 2;

    // float *buf_ptr = buffer.data_ptr<float>();

    // cublas
    const float alpha = 1.0;
    const float beta = 0.0;

    hipblasHandle_t cublasH = at::cuda::getCurrentCUDABlasHandle();

    CUBLAS_CHECK(hipblasSetStream(cublasH, 0));

    if (separate_mid){

        CUBLAS_CHECK(hipblasSetMathMode(cublasH, HIPBLAS_TENSOR_OP_MATH));
        
        CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    out_channel, in_nnz, in_channel, 
                    &alpha, 
                    &weight_ptr[mid_weight_id * in_channel * out_channel], 
                    out_channel, 
                    in_feats_ptr, 
                    in_channel, 
                    &beta, 
                    out_feats_ptr, 
                    out_channel));
    }  

    /********************************************************************/
    // loop over all kernel offsets
    int cur_idx = 0;
    // int stream_id = 0;
    for (int k = 0; k < k_vol; k++){
        int cur_nnz = kernel_nnz.data_ptr<int>()[k];
    
        if (cur_nnz == 0){continue;}

        size_t gridnum_x = DIV_UP(out_channel, 16);
        size_t gridnum_y = DIV_UP(cur_nnz, 64);

        // call batched_gemm
        separate_gemm_fp32_c4<16, 4, 8><<<dim3(gridnum_x, gridnum_y), dim3(4, 16, 1)>>>(
            &kpos_ptr[k], k_vol, in_channel, out_channel, 
            in_feats.data_ptr<float>(), &weight_ptr[k * in_channel * out_channel], out_feats.data_ptr<float>(), 
            &in_map_ptr[cur_idx], &out_map_ptr[cur_idx]
        );

        cur_idx += cur_nnz;
    }
}


void ConvolutionBackward(const at::Tensor out_feats_grad, 
                        const at::Tensor in_feats, 
                        const at::Tensor kernel, 
                        const int ksize_code,
                        const int sum_nnz, 
                        at::Tensor in_feats_grad, 
                        at::Tensor kernel_grad, 
                        const at::Tensor kernel_nnz, 
                        const at::Tensor kernel_pos,
                        const at::Tensor in_map, 
                        const at::Tensor out_map, 
                        const at::Tensor in_csr, 
                        const at::Tensor out_csr, 
                        at::Tensor buffer, 
                        const bool TensorCoreMode
                        ){

    int innz = in_feats.size(0);
    int onnz = out_feats_grad.size(0);
    bool separate_mid = (innz == onnz);
    int in_channel = in_feats.size(1);
    if (in_feats.size(1) != kernel.size(1)) {
        throw std::invalid_argument("Input feature size and kernel size mismatch");
    }
    int out_channel = kernel.size(2);
    int k_vol = kernel.size(0);

    float *ofeats_grad_ptr = out_feats_grad.data_ptr<float>();
    float *in_feats_ptr = in_feats.data_ptr<float>();
    float *weight_ptr = kernel.data_ptr<float>();
    
    float *ifeats_grad_ptr = in_feats_grad.data_ptr<float>();
    float *weight_grad_ptr = kernel_grad.data_ptr<float>();

    int *in_map_ptr = in_map.data_ptr<int>();
    int *out_map_ptr = out_map.data_ptr<int>();
    int *in_csr_ptr = in_csr.data_ptr<int>();
    int *out_csr_ptr = out_csr.data_ptr<int>();

    int *kpos_ptr = kernel_pos.data_ptr<int>();

    int ksx = ksize_code / 94273;
    int ksy = (ksize_code - ksx * 94273) / 311;
    int ksz = ksize_code - ksx * 94273 - ksy * 311;
    int mid_weight_id = (ksx - 1) / 2 * ksy * ksz + 
        (ksy - 1) / 2 * ksz + (ksz - 1) / 2;
    
    float *buf_ptr = buffer.data_ptr<float>();

    int buffer_offset = sum_nnz * in_channel;

    // cublas
    const float alpha = 1.0;
    const float beta = 0.0;
 
    hipblasHandle_t cublasH = at::cuda::getCurrentCUDABlasHandle();
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    // gather for ofeats_grad
    gather_all_input_major_csr_float_4<<<DIV_UP(onnz, 4), 
        dim3(DIV_UP(out_channel, 4), 2, 4), 0, 0>>>(
            onnz, // in_nnz,
            out_channel, // in_channel,
            ofeats_grad_ptr, // in_feats_ptr,
            kpos_ptr, 
            out_csr_ptr, // in_csr_ptr, 
            out_map_ptr, // in_map_ptr,
            &buf_ptr[buffer_offset] // buf_ptr
    );

    /*size_t const block_g = out_channel > PAR_THREAD ? out_channel : PAR_THREAD;
    size_t const grid_g = ((nnz) * (out_channel) + block_g - 1) / block_g;

    gather_all_input_major<<<grid_g, block_g>>>(
            nnz,
            k_vol, 
            sum_nnz,
            kernel_pos_ptr, 
            out_channel,
            ofeats_grad_ptr,
            out_map_ptr,
            out_buffer_ptr
    );*/

    // loop over all kernel offsets: 
    // W^T X {\delta{out_feats}} = {\delta{in_feats}}^T
    int cur_idx = 0;

    for (int i = 0; i < k_vol; i++){

        int cur_nnz = kernel_nnz.data_ptr<int>()[i];
        
        // TODO: put the zero check into the scheduler
        if (cur_nnz == 0){continue;}

        // cublas GEMM for matmul
        if (TensorCoreMode){
            CUBLAS_CHECK(hipblasGemmEx(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                    in_channel, cur_nnz, out_channel, 
                    &alpha, 
                    &weight_ptr[i * in_channel * out_channel], HIP_R_32F, out_channel, 
                    &buf_ptr[buffer_offset + cur_idx * out_channel], HIP_R_32F, out_channel, 
                    &beta, 
                    &buf_ptr[cur_idx * in_channel], HIP_R_32F, in_channel,
                    HIPBLAS_COMPUTE_32F_FAST_16F,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP));
        }
        else{
            CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                    in_channel, cur_nnz, out_channel,
                    &alpha, 
                    &weight_ptr[i * in_channel * out_channel], out_channel, 
                    &buf_ptr[buffer_offset + cur_idx * out_channel], out_channel, 
                    &beta, 
                    &buf_ptr[cur_idx * in_channel], in_channel));
        }
        cur_idx += cur_nnz;
    }

    // scatter for ifeats_grad

    scatter_all_output_major_csr_float<<<DIV_UP(innz, 4), 
        dim3(DIV_UP(in_channel, 4), 4), 0, 0>>>(
            innz, // out_nnz, 
            in_channel, // out_channel, 
            buf_ptr, // &buf_ptr[buffer_offset], 
            kpos_ptr, 
            in_csr_ptr, // out_csr_ptr, 
            in_map_ptr, // out_map_ptr, 
            ifeats_grad_ptr // out_feats_ptr
    );

    /*size_t const block_s = in_channel > PAR_THREAD ? out_channel : PAR_THREAD;
    size_t const grid_s = (nnz * (in_channel) + block_s - 1) / block_s;
        
    scatter_all_output_major<<<grid_s, block_s>>>(
            nnz,
            k_vol, 
            sum_nnz,
            kernel_pos_ptr, 
            in_channel,
            in_buffer_ptr, 
            in_map_ptr,
            ifeats_grad_ptr
    );*/

    // gather for in_feats
    gather_all_input_major_csr_float_4<<<DIV_UP(innz, 4), 
        dim3(DIV_UP(in_channel, 4), 2, 4), 0, 0>>>(
            innz,
            in_channel,
            in_feats_ptr,
            kpos_ptr, 
            in_csr_ptr, 
            in_map_ptr,
            buf_ptr
    );
    /*gather_all_input_major<<<grid_s, block_s>>>(
            nnz,
            k_vol, 
            sum_nnz,
            kernel_pos_ptr, 
            in_channel,
            in_feats_ptr,
            in_map_ptr,
            in_buffer_ptr
    );*/

    // loop over all kernel offsets: 
    // {\delta{out_feats}}^T X in_feats = {\delta{W}}^T
    // reset current idx in the map
    cur_idx = 0;

    for (int i = 0; i < k_vol; i++){

        int cur_nnz = kernel_nnz.data_ptr<int>()[i];
        
        // TODO: put the zero check into the scheduler
        if (cur_nnz == 0){continue;}

        // cublas GEMM for matmul
        if (TensorCoreMode){
            CUBLAS_CHECK(hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                    out_channel, in_channel, cur_nnz, 
                    &alpha, 
                    &buf_ptr[buffer_offset + cur_idx * out_channel], HIP_R_32F, out_channel, 
                    &buf_ptr[cur_idx * in_channel], HIP_R_32F, in_channel, 
                    &beta, 
                    &weight_grad_ptr[i * in_channel * out_channel], HIP_R_32F, out_channel,
                    HIPBLAS_COMPUTE_32F_FAST_16F,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP));
        }
        else{
            CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                    out_channel, in_channel, cur_nnz, 
                    &alpha, 
                    &buf_ptr[buffer_offset + cur_idx * out_channel], out_channel, 
                    &buf_ptr[cur_idx * in_channel], in_channel, 
                    &beta, 
                    &weight_grad_ptr[i * in_channel * out_channel], out_channel));
        }
        cur_idx += cur_nnz;
    }

    // separate computation for center weight w[0, 0, 0]
    // computation for w[0, 0, 0]
    if (separate_mid){
        if (TensorCoreMode){

        CUBLAS_CHECK(hipblasSetMathMode(cublasH, HIPBLAS_TENSOR_OP_MATH));
        
        // W^T X {\delta{out_feats}} = {\delta{in_feats}}^T
        CUBLAS_CHECK(hipblasGemmEx(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                    in_channel, innz, out_channel, 
                    &alpha, 
                    &weight_ptr[mid_weight_id * in_channel * out_channel], HIP_R_32F, out_channel, 
                    ofeats_grad_ptr, HIP_R_32F, out_channel, 
                    &alpha, 
                    ifeats_grad_ptr, HIP_R_32F, in_channel,
                    HIPBLAS_COMPUTE_32F_FAST_16F,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP));

        // {\delta{out_feats}}^T X in_feats = {\delta{W}}^T
        CUBLAS_CHECK(hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                    out_channel, in_channel, innz, 
                    &alpha, 
                    ofeats_grad_ptr, HIP_R_32F, out_channel, 
                    in_feats_ptr, HIP_R_32F, in_channel, 
                    &alpha, 
                    &weight_grad_ptr[mid_weight_id * in_channel * out_channel], HIP_R_32F, out_channel,
                    HIPBLAS_COMPUTE_32F_FAST_16F,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP));
        }
        else{

        // W^T X {\delta{out_feats}} = {\delta{in_feats}}^T
        CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                    in_channel, innz, out_channel,  
                    &alpha, 
                    &weight_ptr[mid_weight_id * in_channel * out_channel], 
                    out_channel, 
                    ofeats_grad_ptr, 
                    out_channel, 
                    &alpha, 
                    ifeats_grad_ptr, 
                    in_channel));
        
        // {\delta{out_feats}}^T X in_feats = {\delta{W}}^T
        CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                    out_channel, in_channel, innz, 
                    &alpha, 
                    ofeats_grad_ptr, 
                    out_channel, 
                    in_feats_ptr, 
                    in_channel, 
                    &alpha, 
                    &weight_grad_ptr[mid_weight_id * in_channel * out_channel], 
                    out_channel));

        }
    }

    return;
}


void gather_with_coded_CSR_wrapper(
                        at::Tensor in_feats, 
                        const at::Tensor kernel_pos, 
                        const at::Tensor in_map, 
                        const at::Tensor in_csr, 
                        at::Tensor buffer){

    int in_nnz = in_feats.size(0);
    // int out_nnz = out_feats.size(0);
    int in_channel = in_feats.size(1);
    // int out_channel = kernel.size(2);
    // int k_vol = kernel.size(0);
   
    int *in_map_ptr = in_map.data_ptr<int>();
    // int *out_map_ptr = out_map.data_ptr<int>();
    int *in_csr_ptr = in_csr.data_ptr<int>();
    // int *out_csr_ptr = out_csr.data_ptr<int>();
    int *kpos_ptr = kernel_pos.data_ptr<int>();
    
    gather_all_input_major_csr_float_4<<<DIV_UP(in_nnz, 4), 
                dim3(DIV_UP(in_channel, 4), 2, 4), 0, 0>>>(
                    in_nnz, in_channel, in_feats.data_ptr<float>(), kpos_ptr, 
                    in_csr_ptr, in_map_ptr, buffer.data_ptr<float>()
    );
}


void scatter_with_coded_CSR_wrapper( 
                        const int buffer_offset, 
                        at::Tensor out_feats, 
                        const at::Tensor kernel_pos, 
                        const at::Tensor out_map, 
                        const at::Tensor out_csr, 
                        at::Tensor buffer){
    
    // int in_nnz = in_feats.size(0);
    int out_nnz = out_feats.size(0);
    // int in_channel = in_feats.size(1);
    // if (in_feats.size(1) != kernel.size(1)) {
    //     throw std::invalid_argument("Input feature size and kernel size mismatch");
    // }
    int out_channel = out_feats.size(1);
    // int k_vol = kernel.size(0);
   
    // int *in_map_ptr = in_map.data_ptr<int>();
    int *out_map_ptr = out_map.data_ptr<int>();
    // int *in_csr_ptr = in_csr.data_ptr<int>();
    int *out_csr_ptr = out_csr.data_ptr<int>();
    int *kpos_ptr = kernel_pos.data_ptr<int>();
    
    scatter_all_output_major_csr_float<<<DIV_UP(out_nnz, 4), 
            dim3(DIV_UP(out_channel, 4), 4), 0, 0>>>(
                out_nnz, out_channel, (buffer.data_ptr<float>() + buffer_offset), kpos_ptr, 
                out_csr_ptr, out_map_ptr, out_feats.data_ptr<float>()
    );
}


void gather_without_coded_CSR_wrapper(
                        const int k_vol, 
                        at::Tensor in_feats, 
                        const at::Tensor kernel_pos, 
                        const at::Tensor in_map, 
                        at::Tensor buffer){

    int in_nnz = in_feats.size(0);
    int in_channel = in_feats.size(1);
    int *in_map_ptr = in_map.data_ptr<int>();
    int *kpos_ptr = kernel_pos.data_ptr<int>();
    
    gather_wo_csr<<<DIV_UP(in_nnz, 2), 
            dim3(DIV_UP(in_channel, 4), 2, 2), 0, 0>>>(
            in_nnz, in_channel, k_vol, in_feats.data_ptr<float>(),
            kpos_ptr, in_map_ptr, buffer.data_ptr<float>()
        );
}


void scatter_without_coded_CSR_wrapper( 
                        const int buffer_offset, 
                        const int k_vol, 
                        at::Tensor out_feats, 
                        const at::Tensor kernel_pos, 
                        const at::Tensor out_map, 
                        at::Tensor buffer){
    
    int out_nnz = out_feats.size(0);
    int out_channel = out_feats.size(1);
    int *out_map_ptr = out_map.data_ptr<int>();
    int *kpos_ptr = kernel_pos.data_ptr<int>();
    
    scatter_wo_csr<<<DIV_UP(out_nnz, 2), 
            dim3(DIV_UP(out_channel, 4), 2), 0, 0>>>(
                out_nnz, out_channel, k_vol, (buffer.data_ptr<float>() + buffer_offset), 
                kpos_ptr, out_map_ptr, out_feats.data_ptr<float>()
        );
}


void map_to_matrix_wrapper(
                        const int nnz, 
                        const int k_vol, 
                        at::Tensor csr, 
                        at::Tensor map, 
                        at::Tensor matrix){
    
    int *csr_ptr = csr.data_ptr<int>();
    int *map_ptr = map.data_ptr<int>();
    int *matrix_ptr = matrix.data_ptr<int>();

    map2matrix<<<dim3(DIV_UP(nnz, 32), 1, 1), dim3(32, 1, 1)>>>(
        nnz, k_vol, csr_ptr, map_ptr, matrix_ptr
    );
}


void torchsparse_gather_wrapper(
                        at::Tensor in_feat, 
                        at::Tensor buffer, 
                        const int kernel_volume, 
                        at::Tensor kpos, 
                        at::Tensor input_mask, 
                        at::Tensor output_mask, 
                        bool transpose, 
                        bool precompute_mid
                        ){

  int n_in_feats = in_feat.size(0);
  int n_in_channels = in_feat.size(1);

  gather_all_kernel_pad_sep_with_mask<float>
            <<<ceil((double)(n_in_feats * n_in_channels) /
                    256), 64>>>(n_in_feats, n_in_channels, kernel_volume,
                      in_feat.data_ptr<float>(),
                      buffer.data_ptr<float>(),
                      kpos.data_ptr<int>(),
                      input_mask.data_ptr<int>(), 
                      output_mask.data_ptr<int>(),
                      transpose, precompute_mid);
}


void torchsparse_scatter_wrapper(
                        at::Tensor out_feat, 
                        at::Tensor buffer, 
                        const int buffer_offset, 
                        const int kernel_volume, 
                        at::Tensor kpos, 
                        at::Tensor input_mask, 
                        at::Tensor output_mask, 
                        bool transpose, 
                        bool precompute_mid
                        ){

  int n_out_feats = out_feat.size(0);
  int n_out_channels = out_feat.size(1);
  
  scatter_all_kernel_pad_sep_with_mask_float<<<
        ceil((double)(n_out_feats * n_out_channels) / 256), 64>>>(
        n_out_feats, n_out_channels, kernel_volume,
        buffer.data_ptr<float>() + buffer_offset, 
        out_feat.data_ptr<float>(),
        kpos.data_ptr<int>(),
        input_mask.data_ptr<int>(),
        output_mask.data_ptr<int>(), 
        transpose, precompute_mid);
}